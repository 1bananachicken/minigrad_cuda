#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstring>
#include "conv2d.cuh"


__global__ void Conv2dForwardKernel(float *X, float *K, float *Y, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride)
{   
    int OH = (H - KH + 1) / stride;
    int OW = (W - KW + 1) / stride;

    int GEMM_M = C_out;
    int GEMM_K = C_in * KH * KW;
    int GEMM_N = N * OH * OW;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < GEMM_M && col < GEMM_N)
    {
        int oc = row;

        int n = col / (OH * OW);
        int j_res = col % (OH * OW);
        int oh = j_res / OW;
        int ow = j_res % OW;

        float accumulator = 0.0f;

        FORLOOP(k, GEMM_K)
        {
            int ic = k / (KH * KW);
            int k_res = k % (KH * KW);
            int kh = k_res / KW;
            int kw = k_res % KW;
            int ih = oh * stride + kh;
            int iw = ow * stride + kw;

            float elem_k = K[IDX2C4D(oc, ic, kh, kw, C_in, KH, KW)];
            float elem_x = X[IDX2C4D(n, ic, ih, iw, C_in, H, W)];

            accumulator += elem_k * elem_x;
        }
        Y[IDX2C2D(row, col, GEMM_N)] = accumulator;
    }
}

__global__ void Conv2dBackwardKernelDX(float *K, float *Dout, float *DX, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride)
{
    int OH = (H - KH + 1) / stride;
    int OW = (W - KW + 1) / stride;

    int GEMM_M = C_in * KH * KW;
    int GEMM_K = C_out;
    int GEMM_N = N * OH * OW;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < GEMM_M && col < GEMM_N)
    {
        int ic = row / (KH * KW);
        int i_res = row % (KH * KW);
        int kh = i_res / KW;
        int kw = i_res % KW;
        int n = col / (OH * OW);
        int j_res = col % (OH * OW);
        int oh = j_res / OW;
        int ow = j_res % OW;
        int ih = oh * stride + kh;
        int iw = ow * stride + kw;

        float accumulator = 0.0f;

        FORLOOP(k, GEMM_K)
        {
            int oc = k;
            float elem_k = K[IDX2C4D(oc, ic, kh, kw, C_in, KH, KW)];
            float elem_dout = Dout[IDX2C4D(n, ic, ih, iw, C_in, H, W)];
            accumulator += elem_k * elem_dout;

        }
        // 这里他妈一定不能写成+=，多线程冲突！！
        atomicAdd(&DX[IDX2C4D(n, ic, ih, iw, C_in, H, W)], accumulator);
    }
}

__global__ void Conv2dBackwardKernelDK(float *X, float *Dout, float *DK, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride)
{
    int OH = (H - KH + 1) / stride;
    int OW = (W - KW + 1) / stride;

    int GEMM_M = C_out;
    int GEMM_K = N * OH * OW;
    int GEMM_N = C_in * KH * KW;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < GEMM_M && col < GEMM_N)
    {
        int oc = row;
        int ic = col / (KH * KW);
        int j_res = col % (KH * KW);
        int kh = j_res / KW;
        int kw = j_res % KW;

        float accumulator = 0.0f;

        FORLOOP(k, GEMM_K)
        {
            int n = k / (OH * OW);
            int k_res = k % (OH * OW);
            int oh = k_res / OW;
            int ow = k_res % OW;
            int ih = oh * stride + kh;
            int iw = ow * stride + kw;

            float elem_dout = Dout[IDX2C4D(n, oc, oh, ow, C_out, OH, OW)];
            float elem_x = X[IDX2C4D(n, ic, ih, iw, C_in, H, W)];

            accumulator += elem_dout * elem_x;
        }
        DK[IDX2C2D(row, col, GEMM_N)] = accumulator;
    }
}

// 数组索引别他妈溢出了
float* Conv2d(float *X, float *kernel, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride, int M, int K, int N_O)
{   
    float *kernel_device, *X_device, *Y_device;
    float *Y_host = (float*)malloc(M * N_O * sizeof(float));
    checkCudaError(hipMalloc(&kernel_device, M * K * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&X_device, N * C_in * H * W * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&Y_device, M * N_O * sizeof(float)), __FILE__, __LINE__);

    checkCudaError(hipMemcpy(kernel_device, kernel, M * K * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
    checkCudaError(hipMemcpy(X_device, X, N * C_in * H * W * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);

    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    dim3 grid_dim((M + BLOCK_DIM - 1) / BLOCK_DIM, (N_O + BLOCK_DIM - 1) / BLOCK_DIM);
    Conv2dForwardKernel<<<grid_dim, block_dim>>>(X_device, kernel_device, Y_device, N, H, W, C_in, C_out, KH, KW, stride);

    checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    checkCudaError(hipDeviceSynchronize(), __FILE__, __LINE__);

    checkCudaError(hipMemcpy(Y_host, Y_device, M * N_O * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    checkCudaError(hipFree(kernel_device), __FILE__, __LINE__);
    checkCudaError(hipFree(X_device), __FILE__, __LINE__);
    checkCudaError(hipFree(Y_device), __FILE__, __LINE__);
    return Y_host;
}

float* Conv2dBackwardDX(float *kernel, float *Dout, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride, int M, int K, int N_O)
{
    float *kernel_device, *Dout_device, *DX_device;
    float *DX_host = (float*)malloc(N * C_in * H * W * sizeof(float));

    checkCudaError(hipMalloc(&kernel_device, M * K * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&Dout_device, K * N_O * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&DX_device, N * C_in * H * W * sizeof(float)), __FILE__, __LINE__);

    checkCudaError(hipMemcpy(kernel_device, kernel, M * K * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
    checkCudaError(hipMemcpy(Dout_device, Dout, K * N_O * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);

    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    dim3 grid_dim((M + BLOCK_DIM - 1) / BLOCK_DIM, (N_O + BLOCK_DIM - 1) / BLOCK_DIM);

    Conv2dBackwardKernelDX<<<grid_dim, block_dim>>>(kernel_device, Dout_device, DX_device, N, H, W, C_in, C_out, KH, KW, stride);

    checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    checkCudaError(hipDeviceSynchronize(), __FILE__, __LINE__);
    
    checkCudaError(hipMemcpy(DX_host, DX_device, N * C_in * H * W * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    checkCudaError(hipFree(kernel_device), __FILE__, __LINE__);
    checkCudaError(hipFree(Dout_device), __FILE__, __LINE__);
    checkCudaError(hipFree(DX_device), __FILE__, __LINE__);

    return DX_host;
}

float* Conv2dBackwardDK(float *X, float *Dout, int N, int H, int W, int C_in, int C_out, int KH, int KW, int stride, int M, int K, int N_O)
{
    float *X_device, *Dout_device, *DK_device;
    float *DK_host = (float*)malloc(M * N_O * sizeof(float));

    checkCudaError(hipMalloc(&X_device, N * C_in * H * W * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&Dout_device, M * K * sizeof(float)), __FILE__, __LINE__);
    checkCudaError(hipMalloc(&DK_device, M * N_O * sizeof(float)), __FILE__, __LINE__);

    checkCudaError(hipMemcpy(X_device, X, N * C_in * H * W * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);
    checkCudaError(hipMemcpy(Dout_device, Dout, M * K * sizeof(float), hipMemcpyHostToDevice), __FILE__, __LINE__);

    dim3 block_dim(BLOCK_DIM, BLOCK_DIM);
    dim3 grid_dim((M + BLOCK_DIM - 1) / BLOCK_DIM, (N_O + BLOCK_DIM - 1) / BLOCK_DIM);

    Conv2dBackwardKernelDK<<<grid_dim, block_dim>>>(X_device, Dout_device, DK_device, N, H, W, C_in, C_out, KH, KW, stride);

    checkCudaError(hipGetLastError(), __FILE__, __LINE__);
    checkCudaError(hipDeviceSynchronize(), __FILE__, __LINE__);
    
    checkCudaError(hipMemcpy(DK_host, DK_device, M * N_O * sizeof(float), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    checkCudaError(hipFree(X_device), __FILE__, __LINE__);
    checkCudaError(hipFree(Dout_device), __FILE__, __LINE__);
    checkCudaError(hipFree(DK_device), __FILE__, __LINE__);

    return DK_host;
}
